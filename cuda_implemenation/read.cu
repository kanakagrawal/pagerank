
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>

using namespace std;

void read(string filename, double** P_sparse, int** row_ind, int** col_ind, int* nnz, int * n)
{
	fstream f(filename.c_str());
	int v, e;
	f >> v;
	f >> e;
	*nnz = e;
	*n = v;
	string dummy;
	int du;
	for(int i = 0; i < v; i++)
	{
		f >> du >> dummy;
	}
	
	*row_ind = new int[e];
	*col_ind = new int[v + 1];
	
	for(int i = 0; i < v + 1; i++)
	{
		(*col_ind)[i] = 0;
	}
	

	*P_sparse = new double[e];
	
	int curLength = 0;
	int curRow, prevRow = 0;
	for(int i = 0; i < *nnz; i++)
	{
		f >> (*row_ind)[i];
		f >> curRow;
		if (curRow != prevRow)
		{
			(*col_ind)[prevRow] = curLength;	
			prevRow = curRow;
		}
		curLength++;
		(*P_sparse)[i] = 1.0;
	}
}

int main()
{
	string filename = "hollins.dat";
	int *row_ind, *col_ind;
	double* P_sparse;
	int nnz, n;
	read(filename, &P_sparse, &row_ind, &col_ind, &nnz, &n);
	cout << n << endl << nnz << endl;
	
	cout << col_ind[0] << endl << col_ind[n-1] << endl;
}
