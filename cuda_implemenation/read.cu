
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>

using namespace std;

void read(string filename, double** P_sparse, int** row_ind, int** col_ind, int* nnz, int * n, int** nnzPerVectorA)
{
	fstream f(filename.c_str());
	int v, e;
	f >> v;
	f >> e;
	*nnz = e;
	*n = v;
	string dummy;
	int du;
	for(int i = 0; i < v; i++)
	{
		f >> du >> dummy;
	}
	
	*row_ind = new int[e];
	*col_ind = new int[v + 1];
	*nnzPerVectorA = new int[v];
	
	for(int i = 0; i < v + 1; i++)
	{
		(*col_ind)[i] = 0;
	}
	

	*P_sparse = new double[e];
	
	int curLengthCumulative = 0;
	int curLength = 0;
	int curRow, prevRow = 0;
	for(int i = 0; i < *nnz; i++)
	{
		f >> (*row_ind)[i];
		(*row_ind)[i]--;
		f >> curRow;
//		curRow--;
		if (curRow != prevRow)
		{
			(*col_ind)[prevRow] = curLengthCumulative;	
			(*nnzPerVectorA)[prevRow - 1] = curLength;
			curLength = 0;
			prevRow = curRow;
		}
		curLength++;
		curLengthCumulative++;
		(*P_sparse)[i] = 1.0;
	}
	(*col_ind)[curRow] = curLengthCumulative;
	(*nnzPerVectorA)[prevRow - 1] = curLength;
}
/*
int main()
{
	string filename = "hollins.dat";
	int *row_ind, *col_ind, *nnzPerVectorA;
	double* P_sparse;
	int nnz, n;
	read(filename, &P_sparse, &row_ind, &col_ind, &nnz, &n, &nnzPerVectorA);
	cout << n << endl << nnz << endl;
	
	cout << col_ind[0] << endl << col_ind[n-1] << endl;
}
*/
