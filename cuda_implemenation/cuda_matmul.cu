#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

void read(string filename, double** P_sparse, int** row_ind, int** col_ind, int* nnz, int * n);

// returns alpha * mat * x
void MatrixMul(double alpha, Matrix *mat, double* d_x_dense, double *d_y_dense)
{
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    const int N = mat->n;                // --- Number of rows and columns
    int nnzA = mat->nnz;                           // --- Number of nonzero elements in dense matrix A

    // --- Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType     (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));  


#ifdef DEBUG
    printf("\nOriginal matrix A in CSR format\n\n");

    printf("\n");
    for (int i = 0; i < nnzA; ++i) printf("P_sparse_ColIndices\n");  
#endif

	double *h_y_dense = (double*)malloc(N * sizeof(double));
	for (int k = 0; k < N; k++)
	{
        h_y_dense[k] = 0.;
    }
	gpuErrchk(hipMemcpy(d_y_dense, h_y_dense, N * sizeof(double), hipMemcpyHostToDevice));
    // Matrix mat = mat->CopyToDevice();

    
    const double beta  = 0.;
    cusparseSafeCall(hipsparseDcsrmv(handle, CUSPARSE_OPERATION_NOT_TRANSPOSE, N, N, nnzA, &alpha, descrA, mat->p, mat->col_ind, mat->row_ind, d_x_dense, 
                                    &beta, d_y_dense));
	gpuErrchk(hipDeviceSynchronize()); 
#ifdef DEBUG
    gpuErrchk(hipMemcpy(h_y_dense, d_y_dense, N * sizeof(double), hipMemcpyDeviceToHost));
    printf("\nResult vector\n\n");
    for (int i = 0; i < N; ++i) printf("h_y[%i] = %f ", i, h_y_dense[i]); printf("\n");
#endif
}

// testing matmul
void mul_test () {
    string filename("data.dat");
    Matrix mat (filename);
    
    // vector<double> x = {1.0, 1.0, 1.0, 1.0};
    // vector<double> y (MatrixMul(1.0, &mat, x));

    // for (int i = 0; i < y.size(); i++)
        // cout << y[i] << endl;
}
