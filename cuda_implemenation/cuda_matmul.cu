#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

void read(string filename, double** P_sparse, int** row_ind, int** col_ind, int* nnz, int * n);

// returns alpha * mat * x
double* MatrixMul(double alpha, Matrix *mat, double* d_x_dense, double *d_y_dense)
{
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    const int N = mat->n;                // --- Number of rows and columns
    int nnzA = mat->nnz;                           // --- Number of nonzero elements in dense matrix A

    // --- Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType     (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));  


#ifdef DEBUG
    printf("\nOriginal matrix A in CSR format\n\n");
    for (int i = 0; i < nnzA; ++i) printf("A[%i] = %f ", i, mat->p[i]); printf("\n");

    printf("\n");
    for (int i = 0; i < (N + 1); ++i) printf("P_sparse_RowIndices[%i] = %i \n", i, mat->col_ind[i]); printf("\n");

    printf("\n");
    for (int i = 0; i < nnzA; ++i) printf("P_sparse_ColIndices[%i] = %i \n", i,mat->row_ind[i]);  
#endif

	double *h_y_dense = (double*)malloc(N * sizeof(double));
	for (int k = 0; k < N; k++)
	{
        h_y_dense[k] = 0.;
    }
	gpuErrchk(hipMemcpy(d_y_dense, h_y_dense, N * sizeof(double), hipMemcpyHostToDevice));
    
    Matrix d_mat = mat->CopyToDevice();

#ifdef DEBUG
    printf("\n");
    for (int i = 0; i < N; ++i) printf("h_x[%i] = %f \n", i, x.data()[i]); printf("\n");
#endif
    
    const double beta  = 0.;
    cusparseSafeCall(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, N, N, nnzA, &alpha, descrA, d_mat.p, d_mat.col_ind, d_mat.row_ind, d_x_dense, 
                                    &beta, d_y_dense));
	gpuErrchk(hipDeviceSynchronize()); 
    
#ifdef DEBUG
    gpuErrchk(hipMemcpy(h_y_dense, d_y_dense, N * sizeof(double), hipMemcpyDeviceToHost));
    printf("\nResult vector\n\n");
    for (int i = 0; i < N; ++i) printf("h_y[%i] = %f ", i, h_y_dense[i]); printf("\n");
#endif
    return d_y_dense;
}

// testing matmul
void mul_test () {
    string filename("data.dat");
    Matrix mat (filename);
    
    // vector<double> x = {1.0, 1.0, 1.0, 1.0};
    // vector<double> y (MatrixMul(1.0, &mat, x));

    // for (int i = 0; i < y.size(); i++)
        // cout << y[i] << endl;
}