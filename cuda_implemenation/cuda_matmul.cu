#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

void read(string filename, double** P_sparse, int** row_ind, int** col_ind, int* nnz, int * n);

int MatrixMul(Matrix mat, vector<int> x, vector<int> *y)
{
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    /**************************/
    /* SETTING UP THE PROBLEM */
    /**************************/
    string filename = "data.dat";
	int *row_ind, *col_ind;
	double* P_sparse;
	int nnz, n;
	read(filename, &P_sparse, &row_ind, &col_ind, &nnz, &n);
    const int N     = n;                // --- Number of rows and columns

    // --- Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;      cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType     (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));  

    int nnzA = nnz;                           // --- Number of nonzero elements in dense matrix A

    printf("\nOriginal matrix A in CSR format\n\n");
    for (int i = 0; i < nnzA; ++i) printf("A[%i] = %f ", i, P_sparse[i]); printf("\n");

    printf("\n");
    for (int i = 0; i < (N + 1); ++i) printf("P_sparse_RowIndices[%i] = %i \n", i, col_ind[i]); printf("\n");

    printf("\n");
    for (int i = 0; i < nnzA; ++i) printf("P_sparse_ColIndices[%i] = %i \n", i, row_ind[i]);  

	double *h_x_dense = (double*)malloc(N * sizeof(double));
	double *h_y_dense = (double*)malloc(N * sizeof(double));
	double *d_A; gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	for (int k = 0; k < N; k++)
	{
        h_x_dense[k] = 1.;
        h_y_dense[k] = 0.;
	}
	double *d_x_dense;  gpuErrchk(hipMalloc(&d_x_dense, N     * sizeof(double)));
	double *d_y_dense; gpuErrchk(hipMalloc(&d_y_dense, N * sizeof(double)));
	gpuErrchk(hipMemcpy(d_x_dense, h_x_dense, N     * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_y_dense, h_y_dense, N * sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_A, P_sparse, nnzA * sizeof(*P_sparse), hipMemcpyHostToDevice));
    
	int *d_row_ind; gpuErrchk(hipMalloc(&d_row_ind, nnzA * sizeof(*d_row_ind)));
	int *d_col_ind; gpuErrchk(hipMalloc(&d_col_ind, (N + 1) * sizeof(*d_col_ind)));
	gpuErrchk(hipMemcpy(d_row_ind, row_ind, nnzA * sizeof(*d_row_ind), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_col_ind, col_ind, (N + 1) * sizeof(*d_col_ind), hipMemcpyHostToDevice));
    

    printf("\n");
    for (int i = 0; i < N; ++i) printf("h_x[%i] = %f \n", i, h_x_dense[i]); printf("\n");

    const double alpha = 1.;
    const double beta  = 0.;
    cusparseSafeCall(hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_TRANSPOSE, N, N, nnzA, &alpha, descrA, d_A, d_col_ind, d_row_ind, d_x_dense, 
                                    &beta, d_y_dense));
	gpuErrchk(hipDeviceSynchronize()); 
    gpuErrchk(hipMemcpy(h_y_dense,           d_y_dense,            N * sizeof(double), hipMemcpyDeviceToHost));

    printf("\nResult vector\n\n");
    for (int i = 0; i < N; ++i) printf("h_y[%i] = %f ", i, h_y_dense[i]); printf("\n");

}
