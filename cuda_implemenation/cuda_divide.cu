#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

double* divide(double* x, double divisor, int n) {
    hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));
    const double alpha = 1/divisor;
    
    double* temp = new double[n];
    for (int i = 0; i < n; i++)
        temp[i] = 0; 
    
    double *d_zero;
    gpuErrchk(hipMalloc(&d_zero, n * sizeof(double)));
    gpuErrchk(hipMemcpy(d_zero, temp, n * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());

    cublasSafeCall(hipblasDaxpy(handle, n, &alpha, x,1,d_zero,1));
    gpuErrchk(hipFree(x));
    gpuErrchk(hipDeviceSynchronize());
    
    delete (temp);
	return d_zero;
}