#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

vector<double> subtract(vector<double> x,vector<double> y)
{
	if (x.size() != y.size()){
		cout<<"Size mismatch for array substraction"<<endl;
		exit(1);
	}

	hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));
	const double alpha = -1.0;

	double *d_x; gpuErrchk(hipMalloc(&d_x, x.size() * sizeof(*d_x)));
	double *d_y; gpuErrchk(hipMalloc(&d_y, y.size() * sizeof(*d_y)));
	gpuErrchk(hipMemcpy(d_x, x.data(), x.size() * sizeof(*d_x), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_y, y.data(), y.size() * sizeof(*d_y), hipMemcpyHostToDevice));

	cublasSafeCall(hipblasDaxpy(handle, x.size(),&alpha, d_y,1,d_x,1));
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(x.data(),d_x, x.size() * sizeof(*d_x), hipMemcpyDeviceToHost));
	return x;
}



void sub_test(){
	vector<double>x(7,300);	
	vector<double>y(7,200);
	cout<<subtract(x,y)[6]<<endl;
}
