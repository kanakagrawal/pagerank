#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

int norm(vector<double> x)
{
	hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));
	
	double *d_x; gpuErrchk(hipMalloc(&d_x, x.size() * sizeof(*d_x)));
	gpuErrchk(hipMemcpy(d_x, x.data(), x.size() * sizeof(*d_x), hipMemcpyHostToDevice));			
	double answer;
	cublasSafeCall(hipblasDasum(handle, x.size(),d_x,1,&answer));
	return answer;
}

int main(){
	vector<double>x(7,300);	
	cout<<norm(x)<<endl;
}
