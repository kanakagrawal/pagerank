#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

int norm(double* d_x, int n)
{
	hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));
	double answer;
	cublasSafeCall(hipblasDasum(handle, n, d_x, 1, &answer));
	gpuErrchk(hipDeviceSynchronize());	
	return answer;
}

void norm_test(){
	vector<double>x(7,300);	
	cout<<norm(x)<<endl;
}
