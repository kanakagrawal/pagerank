#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>
#include <string>
#include <fstream>

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipsparse.h>
using namespace std;
#include "types.cuh"
#include <vector>

double norm(double* d_x, int n)
{
	hipblasHandle_t handle;
	cublasSafeCall(hipblasCreate(&handle));
	double answer;
	cublasSafeCall(hipblasDasum(handle, n, d_x, 1, &answer));
	gpuErrchk(hipDeviceSynchronize());	
	return answer;
}

